
#include <hip/hip_runtime.h>
#include <time.h>
#include <assert.h>
#include <stdio.h>
#include <cudnn.h>
#include <string.h>
#include <stdlib.h>



#define CUDA_CALL(x) do {						\
  hipError_t ____rc = (x);					\
  assert(____rc == hipSuccess);					\
} while (0)

/* Image channels, height, width. */
#define CHANNELS	  3
#define HEIGHT	  1024
#define WIDTH	  1024

/* Tile size. */
#define TILE_WIDTH		32
#define TILE_HEIGHT		32

#define POOL_WIDTH 21
#define POOL_HEIGHT 21

#define DIV_RUP(x, y)	(((x)+(y)-1)/(y))


__global__ void max_pool_kernel(double *global_pointer, double *output_pointer){

	int pad_height = POOL_HEIGHT/2;
	int pad_width = POOL_WIDTH/2;

	int shared_pointer_height = pad_height*2 + blockDim.y;
	int shared_pointer_width = pad_width*2 + blockDim.x;
	__shared__ double shared_pointer[3000];
	int channel_offset = blockIdx.z * HEIGHT * WIDTH;
	
	int global_x_offset = blockDim.x * blockIdx.x;
	int global_y_offset = blockDim.y * blockIdx.y;

	int global_x_val = 0;
	int global_y_val = 0;
	int shared_index = 0;
	int global_index = 0;

	int i = threadIdx.y;	//index of y
	int j = threadIdx.x;	//index of x

	while(i < shared_pointer_height){
		global_y_val = global_y_offset + i - pad_height;
		j = threadIdx.x;
		while(j < shared_pointer_width){
			shared_index = i*shared_pointer_width + j;
			global_x_val = global_x_offset + j - pad_width;
			double global_value = 0.0;
			if(global_y_val < 0 || global_y_val >= HEIGHT || global_x_val < 0 || global_x_val >= WIDTH){
				shared_pointer[shared_index] = 0.0;
			}else{
				global_index = channel_offset + global_y_val*WIDTH + global_x_val;
				global_value = global_pointer[global_index];
				shared_pointer[shared_index] = global_value;
				
			}
			j = j + blockDim.x;
		}
		i = i + blockDim.y;
	}

	 __syncthreads();

	channel_offset = blockIdx.z*HEIGHT*WIDTH;
	double max_val = 0.0;
	int shared_memory_height = blockDim.y + (POOL_HEIGHT/2) * 2;
	int shared_memory_width = blockDim.x + (POOL_WIDTH/2) * 2;

	i = 0;
	j = 0;

	// print_shared_memory(shared_pointer);
	while(i < POOL_HEIGHT){
		j = 0;
		while(j < POOL_WIDTH){
			int x_offset = threadIdx.x + j;
			int y_offset = threadIdx.y + i;
			int shared_index = y_offset*shared_memory_width + x_offset;
			double temp_max = shared_pointer[shared_index];
			if(temp_max > max_val){
				max_val = temp_max;
			}
			j = j + 1;
		}
		i = i + 1;

	}
	int global_x_index = blockDim.x*blockIdx.x + threadIdx.x;
	int global_y_index = blockDim.y*blockIdx.y + threadIdx.y;

	global_index = channel_offset + global_y_index*WIDTH + global_x_index;
	output_pointer[global_index] = max_val;
  	
}

///////////////////////////////////////////////////////////////////////////////
// Create Image in CPU memory
////////////////////////////////////////////////////////////////////////////////
void fill_image(int channels, int height, int width, double *image_pointer)
{
  int image_memory_size = channels*height*width*sizeof(double);
  memset(image_pointer, 0, image_memory_size);
  for(int k = 0; k < channels; k++){
  	for(int i = 0; i < height; i++){
  		for(int j = 0; j < width; j++){
  			int index = i*WIDTH + j + k*WIDTH*HEIGHT;
  			image_pointer[index] = k*(i+j);
  		}
  	}
  }
}

void validate_image_data(int channels, int height, int width, double *image_pointer){
	double sum = 0.0;

	for(int k = 0; k < channels; k++)
		for(int i = 0; i < height; i++){
  			for(int j = 0; j < width; j++){
  			{
  				int index = i*WIDTH + j + k*WIDTH*HEIGHT;
  				sum = sum + image_pointer[index];
  			}
  		}
  	}
  	printf("Check sum value is %lf \n",sum);
  	if(sum == 3218079744.0){
  		printf("Check sum of image validated \n");
  	}
  	else{
  		printf("Check sum is wrong.\n",sum);
  		printf("Exiting program \n");
  		exit(0);
  	}
}

void print_max_pool_checksum(int channels, int height, int width, double *output_pointer){
	double sum = 0.0;
	for(int k = 0; k < channels; k++)
		for(int i = 0; i < height; i++){
  			for(int j = 0; j < width; j++){
  			{
  				int index = i*WIDTH + j + k*WIDTH*HEIGHT;
  				sum = sum + output_pointer[index];
  			}
  		}
  	}
  	printf("The checksum after the max_pool is %lf \n",sum);
}
void check_on_cpu(double *image_pointer, double *output_pointer){
	int pooling_height = POOL_HEIGHT;
	int pooling_width = POOL_WIDTH;

  	int index = 0;
  	int pad_height = pooling_height/2;
  	int pad_width = pooling_width/2;
  	int output_index = 0;
  	for(int c = 0; c < CHANNELS; c++){
  		int offset = c*HEIGHT*WIDTH;
  		for(int i = 0; i< HEIGHT; i++){
	  		for(int j = 0; j< WIDTH; j++){
	  			int start_i = i;
	  			int start_j = j;

	  			int max_val = 0;
	  			for(int k = start_i - pad_height; k <= start_i+pad_height; k++){
	  				for(int l = start_j - pad_width; l <= start_j+pad_width; l++){
	  					if(k >= 0 && k < HEIGHT && l >= 0 && l < WIDTH){
	  						index = offset + k*WIDTH + l;
	  						if(image_pointer[index] > max_val){
	  							max_val = image_pointer[index];
	  						}
	  					}
	  				}
	  			}
	  			output_index = offset + i*WIDTH + j;
	  			output_pointer[output_index] = max_val;
	  		}
	  	}
  	}
}

int main(int ac, char *av[]){
	int image_size = CHANNELS*HEIGHT*WIDTH*sizeof(double);
	double *gpu_image_pointer, *gpu_output_pointer;
	double *image_pointer, *output_pointer, *cpu_output_pointer;

	image_pointer = (double *) malloc(image_size);
	output_pointer = (double *) malloc(image_size);
	cpu_output_pointer = (double *) malloc(image_size);
	memset(output_pointer, 0, image_size);
	memset(cpu_output_pointer, 0, image_size);
  	fill_image(CHANNELS, HEIGHT, WIDTH, image_pointer);
  	validate_image_data(CHANNELS, HEIGHT, WIDTH, image_pointer);

  	CUDA_CALL(hipMalloc(&gpu_image_pointer, image_size));
  	CUDA_CALL(hipMalloc(&gpu_output_pointer, image_size));
  	CUDA_CALL(hipMemcpy(gpu_image_pointer, image_pointer, image_size, hipMemcpyHostToDevice));
  	hipDeviceSynchronize();

  	dim3 image_block_vector(TILE_WIDTH, TILE_HEIGHT);
  	dim3 image_grid_vector(DIV_RUP(WIDTH, TILE_WIDTH), DIV_RUP(HEIGHT, TILE_HEIGHT), CHANNELS);

  	max_pool_kernel<<<image_grid_vector, image_block_vector>>>(gpu_image_pointer, gpu_output_pointer);
    hipDeviceSynchronize();
    hipMemcpy(output_pointer, gpu_output_pointer, image_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    print_max_pool_checksum(CHANNELS, HEIGHT, WIDTH, output_pointer);
    check_on_cpu(image_pointer, cpu_output_pointer);
    print_max_pool_checksum(CHANNELS, HEIGHT, WIDTH, cpu_output_pointer);

  	
}



